#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

/*
配列の形

node 0:state
     1:height
     2:out_root_id
     3:in_root_id
     4:source,sinkにつながっているかのflg

edge 0:flow
     1:reverse_id
     2:in_node_id
     3:out_node_id
     4:in_link_id
     5:out_link_id
     6:id
*/

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess) 
    {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}
  
#define NODE 5
#define EDGE 7

#define NADR(NUM, ID, EL) (NUM * EL + ID)
#define EADR(NUM, ID, EL) ((NUM * 2) * EL + ID)

//node
#define STATE 0
#define HEIGHT 1
#define OUT_ROOT 2
#define IN_ROOT 3
#define ST_FLG 4

//edge
#define FLOW 0
#define ROUTE 1
#define REVERSE 2
#define IN_NODE 3
#define IN_LINK 4
#define OUT_NODE 5
#define OUT_LINK 6

int *N_NUM;
int *E_NUM;
int *n_table;
int *e_table;
int *source;
int *sink;

void link(int e_table, int from, int to, int r_edge, int flow);
int load(FILE *fp);

__global__ void node_reset(int *n_table, int *e_table, int *source, int *sink, int *N_NUM, int *E_NUM, int *e_flg) {
  int total_id = blockDim.x * blockIdx.x + threadIdx.x;
  e_flg[total_id] = 0;
  if (total_id >= N_NUM[0]) return;
  if (total_id == source[0]) {
    n_table[NADR(N_NUM[0], total_id, STATE)] = 1;
    n_table[NADR(N_NUM[0], total_id, HEIGHT)] = 0;
    return;
  }
  else if (total_id == sink[0]) {
    n_table[NADR(N_NUM[0], total_id, STATE)] = 2;
    n_table[NADR(N_NUM[0], total_id, HEIGHT)] = 0;
    return;
  }
  else {
    if (n_table[NADR(N_NUM[0], total_id, ST_FLG)] == -1) {
      n_table[NADR(N_NUM[0], total_id, HEIGHT)] = N_NUM[0];
      n_table[NADR(N_NUM[0], total_id, STATE)] = 0;
      return;
    }
    else {
      int f_flg = n_table[NADR(N_NUM[0], total_id, ST_FLG)];
      if (e_table[EADR(E_NUM[0], f_flg, FLOW)] > 0) {
	int in_node = e_table[EADR(E_NUM[0], f_flg, IN_NODE)];
	n_table[NADR(N_NUM[0], total_id, HEIGHT)] = 1;
	if (in_node == source[0]) n_table[NADR(N_NUM[0], total_id, STATE)] = 1;
	else n_table[NADR(N_NUM[0], total_id, STATE)] = 2;
      }
      else {
	n_table[NADR(N_NUM[0], total_id, HEIGHT)] = N_NUM[0];
	n_table[NADR(N_NUM[0], total_id, STATE)] = 0;
	n_table[NADR(N_NUM[0], total_id, ST_FLG)] = -1;
      }
    }
  }
}

__global__ void trace_cu(int *n_table, int *e_table, int *flg3, int *que3, int *N_NUM, int *E_NUM, int *cnt, int *flg1, int *e_flg) {
  int node_id = blockDim.x * blockIdx.x + threadIdx.x;
  if (node_id >= N_NUM[0]) return;

  __shared__ int n_num[1];
  n_num[0] = N_NUM[0];
  __shared__ int e_num[1];
  e_num[0] = E_NUM[0];

  if (n_table[NADR(n_num[0], node_id, HEIGHT)] != cnt[0]) return;
  int state, look_n, root, link;
  if (n_table[NADR(n_num[0], node_id, STATE)] == 1) {
    state = 1;
    look_n = OUT_NODE;
    root = OUT_ROOT;
    link = OUT_LINK;
  }
  else {
    state = 2;
    look_n = IN_NODE;
    root = IN_ROOT;
    link = IN_LINK;
  }
  int old;
  int lok_node;
  int edge_id = n_table[NADR(n_num[0], node_id, root)];
  for (;;) {
    for (;;) {
      if (edge_id == -1) return;
      if (e_table[EADR(e_num[0], edge_id, FLOW)] > 0) break;
      edge_id = e_table[EADR(e_num[0], edge_id, link)];
    }
    lok_node = e_table[EADR(e_num[0], edge_id, look_n)];
    if (n_table[NADR(n_num[0], lok_node, STATE)] == 0) {
      old = atomicExch(&e_flg[lok_node], 1);
      if (old == 0) {
	n_table[NADR(n_num[0], lok_node, STATE)] = state;
	n_table[NADR(n_num[0], lok_node, HEIGHT)] = n_table[NADR(n_num[0], node_id, HEIGHT)] + 1;
	flg1[0] = 1;
      }
    }
    else if (n_table[NADR(n_num[0], lok_node, STATE)] != state) {
      old = atomicAdd(&(flg3[0]), 1);
      que3[old] = edge_id;
    }
    edge_id = e_table[EADR(e_num[0], edge_id, link)];
  }
}

__device__ void flow_t(int *e_table, int flow, int pre_edge, int edge, int N_NUM, int E_NUM, int tag, int *e_flg) {
  atomicSub(&e_table[EADR(E_NUM, edge, FLOW)], flow);
  int reverse = e_table[EADR(E_NUM, edge, REVERSE)];
  atomicAdd(&e_table[EADR(E_NUM, reverse, FLOW)], flow);

  for (;;) {
    atomicSub(&e_table[EADR(E_NUM, pre_edge, FLOW)], flow);
    reverse = e_table[EADR(E_NUM, pre_edge, REVERSE)];
    atomicAdd(&e_table[EADR(E_NUM, reverse, FLOW)], flow);
    if (pre_edge == tag) break;
    pre_edge = e_table[EADR(E_NUM, pre_edge, ROUTE)];
  }
}

__global__ void aug_cu(int *n_table, int *e_table, int *flg1, int *que1, int *source, int *sink, int *e_flg, int *flow_sum, int *N_NUM, int *E_NUM) {
  int total_id = blockDim.x * blockIdx.x + threadIdx.x;
  if (total_id >= flg1[0]) return;
  int tag = que1[total_id];
  int old = atomicExch(&e_flg[tag], 1);
  if (old != 0) return;
  int pre_edge = tag;

  __shared__ int n_num[1];
  n_num[0] = N_NUM[0];
  __shared__ int e_num[1];
  e_num[0] = E_NUM[0];

  int node = e_table[EADR(e_num[0], tag, IN_NODE)];
  int flow = e_table[EADR(e_num[0], tag, FLOW)];
  int root, look_n, link;
  int state = 1;
  int height = n_table[NADR(N_NUM[0], node, HEIGHT)] - 1;
  for (;;) {
    if (state == 1) {
      root = IN_ROOT;
      look_n = IN_NODE;
      link = IN_LINK;
    }
    else {
      root = OUT_ROOT;
      look_n = OUT_NODE;
      link = OUT_LINK;
    }
    int edge_id = n_table[NADR(n_num[0], node, root)];
    int flow1;
    int lok_node;
    for (;;) {
      if (edge_id == -1) return;
      flow1 = e_table[EADR(e_num[0], edge_id, FLOW)];
      lok_node = e_table[EADR(e_num[0], edge_id, look_n)];
      if (flow1 > 0) {
	if (n_table[NADR(n_num[0], lok_node, HEIGHT)] == height) {
	  if (n_table[NADR(n_num[0], lok_node, STATE)] == state) {
	    old = atomicExch(&e_flg[edge_id], 1);
	    if (old == 0) {
	      flow = ((flow > flow1) ? flow1 : flow);
	      break;
	    }
	  }
	}
      }
      edge_id = e_table[EADR(e_num[0], edge_id, link)];
    }
    if (lok_node == sink[0]) {
      flow_t(e_table, flow, pre_edge, edge_id, n_num[0], e_num[0], tag, e_flg);
      atomicAdd(&flow_sum[0], flow);
      return;
    }
    else if (lok_node == source[0]) {
      node = e_table[EADR(e_num[0], tag, OUT_NODE)];
      height = n_table[NADR(n_num[0], node, HEIGHT)] - 1;
      state = 2;
    }
    else {
      node = lok_node;
      height--;
    }
    e_table[EADR(e_num[0], edge_id, ROUTE)] = pre_edge;
    pre_edge = edge_id;
  }
}

__global__ void flg_reset(int *flg) {
  flg[0] = 0;
}

__global__ void cnt_add(int *cnt) {
  cnt[0]++;
}

int main(int argc, char **argv) {
  if (argc != 2) {
    printf("Usage: ./bk.exe file.inp\n");
    return 1;
  }
  FILE *fp = fopen(argv[1], "r");
  if (fp == NULL) {
    printf("Can't open file [%s]\n", argv[1]);
    return 1;
  }
  int max = load(fp);
  fclose(fp);

  if (max == -1) return 0;//ファイルの形式がおかしい時return

  clock_t start, end;//時間計測用

  //hostの変数確保
  int *flg1, *flg3;
  gpuErrchk ( hipHostMalloc(&flg1, sizeof(int)) );
  gpuErrchk ( hipHostMalloc(&flg3, sizeof(int)) );
  int *flow_sum;
  gpuErrchk ( hipHostMalloc(&flow_sum, sizeof(int)) );

  size_t C_SIZE = sizeof(int);
  size_t N_SIZE = sizeof(int) * N_NUM[0] * NODE;
  size_t E_SIZE = sizeof(int) * E_NUM[0] * EDGE * 2;

  //deviceの変数確保
  int *DN_NUM;
  int *DE_NUM;
  int *nd_table;
  int *ed_table;
  int *d_flg1, *d_flg3;
  int *d_que3;
  int *d_source, *d_sink;
  int *dflow_sum;
  int *de_flg;
  int *d_cnt;

  gpuErrchk( hipMalloc((void**)&DN_NUM, C_SIZE) );
  gpuErrchk( hipMalloc((void**)&DE_NUM, C_SIZE) );
  gpuErrchk( hipMalloc((void**)&d_flg1, C_SIZE) );
  gpuErrchk( hipMalloc((void**)&d_flg3, C_SIZE) );
  gpuErrchk( hipMalloc((void**)&d_que3, sizeof(int) * E_NUM[0]) );
  gpuErrchk( hipMalloc((void**)&nd_table, N_SIZE) );
  gpuErrchk( hipMalloc((void**)&ed_table, E_SIZE) );
  gpuErrchk( hipMalloc((void**)&d_source, C_SIZE) );
  gpuErrchk( hipMalloc((void**)&d_sink, C_SIZE) );
  gpuErrchk( hipMalloc((void**)&dflow_sum, C_SIZE) );
  gpuErrchk( hipMalloc((void**)&de_flg, sizeof(int) * E_NUM[0] * 2) );
  gpuErrchk (hipMalloc((void**)&d_cnt, sizeof(int)) );

  //deviceへのコピー
  gpuErrchk( hipMemcpy(nd_table, n_table, N_SIZE, hipMemcpyHostToDevice) );
  gpuErrchk( hipMemcpy(ed_table, e_table, E_SIZE, hipMemcpyHostToDevice) );
  gpuErrchk( hipMemcpy(d_source, source, C_SIZE, hipMemcpyHostToDevice) );
  gpuErrchk( hipMemcpy(d_sink, sink, C_SIZE, hipMemcpyHostToDevice) );
  gpuErrchk( hipMemcpy(DN_NUM, N_NUM, C_SIZE, hipMemcpyHostToDevice) );
  gpuErrchk( hipMemcpy(DE_NUM, E_NUM, C_SIZE, hipMemcpyHostToDevice) );

  //計測開始
  start = clock();

  flg_reset<<<1, 1>>>(dflow_sum);//flow_sumを初期化
  for (;;) {
    flg_reset<<<1, 1>>>(d_flg3);
    flg_reset<<<1, 1>>>(d_cnt);
    node_reset<<<(N_NUM[0] / 32) + 1, 32>>>(nd_table, ed_table, d_source, d_sink, DN_NUM, DE_NUM, de_flg);
    gpuErrchk( hipDeviceSynchronize() );
    //growth stage
    for (;;) {
      flg_reset<<<1, 1>>>(d_flg1);
      cnt_add<<<1, 1>>>(d_cnt);
      gpuErrchk( hipDeviceSynchronize() );
      trace_cu<<<((N_NUM[0]) / 128) + 1, 128>>>(nd_table, ed_table, d_flg3, d_que3, DN_NUM, DE_NUM, d_cnt, d_flg1, de_flg);
      gpuErrchk( hipDeviceSynchronize() );
      gpuErrchk( hipMemcpy(flg1, d_flg1, C_SIZE, hipMemcpyDeviceToHost) );
      if (flg1[0] == 0) break;
    }

    //ぶつかったエッジがあったかを確認
    gpuErrchk( hipMemcpy(flg3, d_flg3, sizeof(int), hipMemcpyDeviceToHost) );
    if (flg3[0] == 0) break;

    //augmentation stage
    for (int i = 0; i < 3; i++) {
      gpuErrchk( hipMemset((void**)de_flg, 0, sizeof(int) * E_NUM[0] * 2) );
      aug_cu<<<flg3[0] / 32 + 1, 32>>>(nd_table, ed_table, d_flg3, d_que3, d_source, d_sink, de_flg, dflow_sum, DN_NUM, DE_NUM);
      gpuErrchk( hipDeviceSynchronize() );
    }

    gpuErrchk( hipMemcpy(flow_sum, dflow_sum, sizeof(int), hipMemcpyDeviceToHost) );
    printf("current_flow : %d\n", flow_sum[0]);
  }

  printf("flow_sum : %d\n", flow_sum[0]);
  end = clock();
  //計測終了
  printf("time:%.2f[s]\n", (double)(end - start) / CLOCKS_PER_SEC);

  hipFree(DN_NUM);
  hipFree(DE_NUM);
  hipFree(nd_table);
  hipFree(ed_table);
  hipFree(d_flg1);
  hipFree(d_flg3);
  hipFree(d_que3);
  hipFree(d_source);
  hipFree(d_sink);
  hipFree(dflow_sum);
  hipFree(d_cnt);
  hipFree(de_flg);

  hipHostFree(flg1);
  hipHostFree(flg3);
  hipHostFree(n_table);
  hipHostFree(e_table);
  hipHostFree(source);
  hipHostFree(sink);
  hipHostFree(flow_sum);
  hipHostFree(N_NUM);
  hipHostFree(E_NUM);

  return 0;
}

void link(int *n_table, int *e_table, int from, int to, int flow, int edge_id, int *count) {
  int edge = n_table[NADR(N_NUM[0], from, OUT_ROOT)];
  for (;;) {
    if (edge == -1) break;
    if (e_table[EADR(E_NUM[0], edge, OUT_NODE)] == to) {
      e_table[EADR(E_NUM[0], edge, FLOW)] += flow;
      count[0]++;
      return;
    }
    edge = e_table[EADR(E_NUM[0], edge, OUT_LINK)];
  }
  //正のエッジ
  e_table[EADR(E_NUM[0], edge_id, REVERSE)] = edge_id + 1;
  e_table[EADR(E_NUM[0], edge_id, FLOW)] = flow;
  e_table[EADR(E_NUM[0], edge_id, IN_NODE)] = from;
  e_table[EADR(E_NUM[0], edge_id, OUT_NODE)] = to;
  e_table[EADR(E_NUM[0], edge_id, OUT_LINK)] = n_table[NADR(N_NUM[0], from, OUT_ROOT)];
  n_table[NADR(N_NUM[0], from, OUT_ROOT)] = edge_id;
  e_table[EADR(E_NUM[0], edge_id, IN_LINK)] = n_table[NADR(N_NUM[0], to, IN_ROOT)];
  n_table[NADR(N_NUM[0], to, IN_ROOT)] = edge_id;
  if (from == source[0]) {
    n_table[NADR(N_NUM[0], to, ST_FLG)] = edge_id;
  }
  if (to == sink[0]) {
    n_table[NADR(N_NUM[0], from, ST_FLG)] = edge_id;
  }

  //逆(reverse)のエッジ
  e_table[EADR(E_NUM[0], (edge_id + 1), REVERSE)] = edge_id;
  e_table[EADR(E_NUM[0], (edge_id + 1), FLOW)] = 0;
  e_table[EADR(E_NUM[0], (edge_id + 1), IN_NODE)] = to;
  e_table[EADR(E_NUM[0], (edge_id + 1), OUT_NODE)] = from;
  e_table[EADR(E_NUM[0], (edge_id + 1), OUT_LINK)] = n_table[NADR(N_NUM[0], to, OUT_ROOT)];
  n_table[NADR(N_NUM[0], to, OUT_ROOT)] = edge_id + 1;
  e_table[EADR(E_NUM[0], (edge_id + 1), IN_LINK)] = n_table[NADR(N_NUM[0], from, IN_ROOT)];
  n_table[NADR(N_NUM[0], from, IN_ROOT)] = edge_id + 1;
}

int load(FILE *fp) {
  int max = 0;
  char s1[10], s2[10];
  gpuErrchk( hipHostMalloc(&N_NUM, sizeof(int)) );
  gpuErrchk( hipHostMalloc(&E_NUM, sizeof(int)) );
  int result = fscanf(fp, "%s %s %d %d\n", s1, s2, &N_NUM[0], &E_NUM[0]);
  if (result == EOF) return -1;
  gpuErrchk( hipHostMalloc(&n_table, sizeof(int) * N_NUM[0] * NODE) );
  gpuErrchk( hipHostMalloc(&e_table, sizeof(int) * E_NUM[0] * EDGE * 2) );

  for (int i = 0; i < N_NUM[0]; i++) {
    n_table[NADR(N_NUM[0], i, OUT_ROOT)] = -1;
    n_table[NADR(N_NUM[0], i, IN_ROOT)] = -1;
    n_table[NADR(N_NUM[0], i, ST_FLG)] = -1;
  }
  for (int i = 0; i < E_NUM[0] * 2; i++) {
    e_table[EADR(E_NUM[0], i, REVERSE)] = -1;
    e_table[EADR(E_NUM[0], i, IN_NODE)] = -1;
    e_table[EADR(E_NUM[0], i, OUT_NODE)] = -1;
    e_table[EADR(E_NUM[0], i, IN_LINK)] = -1;
    e_table[EADR(E_NUM[0], i, OUT_LINK)] = -1;
  }
  int t;
  result = fscanf(fp, "%s %d %s\n", s1, &t, s2);
  if (result == EOF) return -1;
  hipHostMalloc(&source, sizeof(int));
  source[0] = t;
  result = fscanf(fp, "%s %d %s\n", s1, &t, s2);
  if (result == EOF) return -1;
  hipHostMalloc(&sink, sizeof(int));
  sink[0] = t;

  int *count;
  count = new int[1]();
  for (int i = 0; i < E_NUM[0]; i++) {
    int from, to, flow;
    result = fscanf(fp, "%s %d %d %d\n", s1, &from, &to, &flow);
    if (result == EOF) break;
    if (flow > max) max = flow;
    link(n_table, e_table, from, to, flow, (i - count[0]) * 2, count);
  }
  delete[] count;

  return max;
}
